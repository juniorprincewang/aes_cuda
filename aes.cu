#include "hip/hip_runtime.h"
#include "aes.h"

const char *file_path = "plaintext.txt";

//generate round keys from initial key
void expand_key(uint8_t *key, uint8_t *rkey){
  uint32_t i,j,k;
  uint8_t tempa[4];
  uint32_t nround = 10;

  //first round key is just the key
  for(i = 0; i < 4; ++i){
    rkey[4*i + 0] = key[4*i + 0];
    rkey[4*i + 1] = key[4*i + 1];
    rkey[4*i + 2] = key[4*i + 2];
    rkey[4*i + 3] = key[4*i + 3];
  }

  for(i = 4; i < 4*(nround + 1); ++i){
    for(j = 0; j < 4; ++j){
      tempa[j] = rkey[(i-1)*4 + j];
    }

    if(i % 4 == 0){
      //rotate 4 bytes in word
      k = tempa[0];
      tempa[0] = tempa[1];
      tempa[1] = tempa[2];
      tempa[2] = tempa[3];
      tempa[3] = k;


      tempa[0] = sbox[tempa[0]];
      tempa[1] = sbox[tempa[1]];
      tempa[2] = sbox[tempa[2]];
      tempa[3] = sbox[tempa[3]];
  
      tempa[0] = tempa[0] ^ rcon[i/4];

    }

    rkey[4*i + 0] = rkey[4*(i-4) + 0] ^ tempa[0];
    rkey[4*i + 1] = rkey[4*(i-4) + 1] ^ tempa[1];
    rkey[4*i + 2] = rkey[4*(i-4) + 2] ^ tempa[2];
    rkey[4*i + 3] = rkey[4*(i-4) + 3] ^ tempa[3];

  } 

}

//XOR round key with block(1 block per thread)
__device__ void add_round_key(uint8_t *block, uint8_t *key, uint32_t offset){
  //word size traversal
  uint32_t *b = (uint32_t *)block;
  uint32_t *k = (uint32_t *)key;
  for(int i = 0; i < 4; ++i){
    b[offset/4 + i] = b[offset/4 + i] ^ k[i];
  }  
}

//substitute block int sbox (1 block per thread)
__device__ void sub_bytes(uint8_t *block, uint32_t offset){
  for(int i = 0; i < 16; ++i){
    block[offset + i] = dsbox[block[offset + i]];
  }
}

//substitute block int sbox (1 block per thread)
__device__ void inv_sub_bytes(uint8_t *block, uint32_t offset){
  for(int i = 0; i < 16; ++i){
    block[offset + i] = disbox[block[offset + i]];
  }
}


//mix columns by taking linear combinations in the field (1 block per thread)
__device__ void mix_columns(uint8_t *block, uint32_t offset){
  for(int i = 0; i < 4; ++i){ //iterate over columns
    uint8_t a[4];
    uint8_t b[4]; 
    uint8_t h;
  
    for(int j = 0; j < 4; ++j){
      a[j] = block[offset + 4*i + j];
      h = (uint8_t)((int8_t)a[j] >> 7);
      b[j] = a[j] << 1;
      b[j] ^= 0x1b & h;
    } 

    block[offset + 4*i + 0] = b[0] ^ a[3] ^ a[2] ^ b[1] ^ a[1];
    block[offset + 4*i + 1] = b[1] ^ a[0] ^ a[3] ^ b[2] ^ a[2];
    block[offset + 4*i + 2] = b[2] ^ a[1] ^ a[0] ^ b[3] ^ a[3];
    block[offset + 4*i + 3] = b[3] ^ a[2] ^ a[1] ^ b[0] ^ a[0]; 

  }
}

//mix columns by taking linear combinations in the field (1 block per thread)
/**
f(x) = 11 * x^3 + 13 * x^2 + 9 * x +  14
*/
__device__ void inv_mix_columns(uint8_t *block, uint32_t offset){

	for(int i = 0; i < 4; ++i){ //iterate over columns
		uint16_t t;
		uint8_t a[4];
		uint8_t a2[4];
		uint8_t a4[4];
		uint8_t a8[4];
		a[0] = block[offset + 4*i];
		a[1] = block[offset + 4*i + 1];
		a[2] = block[offset + 4*i + 2];
		a[3] = block[offset + 4*i + 3];
		a2[0] = gmul2(a[0]);
		a2[1] = gmul2(a[1]);
		a2[2] = gmul2(a[2]);
		a2[3] = gmul2(a[3]);
		a4[0] = gmul4(a[0]);
		a4[1] = gmul4(a[1]);
		a4[2] = gmul4(a[2]);
		a4[3] = gmul4(a[3]);
		a8[0] = gmul8(a[0]);
		a8[1] = gmul8(a[1]);
		a8[2] = gmul8(a[2]);
		a8[3] = gmul8(a[3]);

		block[offset + 4*i + 0] = a8[0] ^ a4[0] ^ a2[0] ^ a8[1] ^ a2[1] ^ a[1] ^ a8[2] ^ a4[2] ^ a[2] ^ a8[3] ^ a[3];
		block[offset + 4*i + 1] = a8[0] ^ a[0] ^ a8[1] ^ a4[1] ^ a2[1] ^ a8[2] ^ a2[2] ^ a[2] ^ a8[3] ^ a4[3] ^ a[3];
		block[offset + 4*i + 2] = a8[0] ^ a4[0] ^ a[0] ^ a8[1] ^ a[1] ^ a8[2] ^ a4[2] ^ a2[2] ^ a8[3] ^ a2[3] ^ a[3];
		block[offset + 4*i + 3] = a8[0] ^ a2[0] ^ a[0] ^ a8[1] ^ a4[1] ^ a[1] ^ a8[2] ^ a[2] ^ a8[3] ^ a4[3] ^ a2[3];
	}
}


//shift rows left by 0,1,2,3 bytes respectively (1 block per thread)
__device__ void shift_rows(uint8_t *sblock, uint32_t offset){
  uint8_t tmp;

  uint8_t *block = sblock + offset; 

  //row 0 remains unshifted

  //shift row 1 left by 1
  tmp = block[1];
  block[1] = block[5];
  block[5] = block[9];
  block[9] = block[13];
  block[13] = tmp;

  //shift row 2 letf by 2
  tmp = block[2];
  block[2] = block[10];
  block[10] = tmp;

  tmp = block[6];
  block[6] = block[14];
  block[14] = tmp;

  //shift row 3 left by 3
  tmp = block[3];
  block[3] = block[15];
  block[15] = block[11];
  block[11] = block[7];
  block[7] = tmp;
}


//shift rows right by 0,1,2,3 bytes respectively (1 block per thread)
__device__ void inv_shift_rows(uint8_t *sblock, uint32_t offset){
  uint8_t tmp;
  uint8_t *block = sblock + offset; 

  //row 0 remains unshifted

  //shift row 1 right by 1
  tmp = block[13];
  block[13] = block[9];
  block[9] 	= block[5];
  block[5] = block[1];
  block[1] = tmp;

  //shift row 2 right by 2
  tmp = block[10];
  block[10] = block[2];
  block[2] = tmp;

  tmp = block[14];
  block[14] = block[6];
  block[6] = tmp;

  //shift row 3 right by 3
  tmp = block[3];
  block[3] = block[7];
  block[7] = block[11];
  block[11] = block[15];
  block[15] = tmp;
}

//aes 128 encryption with expanded key supplied
//implemented as basic byte algorithm (naive)
//operates on one block per thread
__device__ void encrypt(uint8_t *block, uint8_t *rkey, uint32_t offset){
	add_round_key(block, rkey, offset);
	for(int i = 1; i < 10; ++i){
		sub_bytes(block, offset);
		shift_rows(block, offset);
		mix_columns(block, offset);
		add_round_key(block, rkey + 16*i, offset);
	}
	sub_bytes(block, offset);
	shift_rows(block, offset);
	add_round_key(block, rkey + 160, offset);
}

__global__ void encrypt_one_block(uint8_t *block, uint8_t *rkey, uint32_t numblock)
{
	int bindex = blockIdx.x * blockDim.x + threadIdx.x;
	int offset = bindex * 16;
	if(bindex >= numblock) return;
	// printf("bindex = %d\n", bindex);
	encrypt(block, rkey, offset);
}

/***********************************************************************/
/* decrypt */
/***********************************************************************/

//aes 128 encryption with expanded key supplied
//implemented as basic byte algorithm (naive)
//operates on one block per thread
__device__ void decrypt(uint8_t *block, uint8_t *rkey, uint32_t offset){
	add_round_key(block, rkey+ 160, offset);
	for(int i = 9; i >=1 ; --i){
		inv_shift_rows(block, offset);
		inv_sub_bytes(block, offset);
		add_round_key(block, rkey + 16*i, offset);
		inv_mix_columns(block, offset);
	}
	inv_shift_rows(block, offset);
	inv_sub_bytes(block, offset);
	add_round_key(block, rkey, offset);
}

__global__ void decrypt_one_block(uint8_t *block, uint8_t *rkey, uint32_t numblock)
{
	int bindex = blockIdx.x * blockDim.x + threadIdx.x;
	int offset = bindex * 16;
	if(bindex >= numblock) return;
	// printf("bindex = %d\n", bindex);
	decrypt(block, rkey, offset);
}

void encrypt_cuda(uint8_t *data, uint8_t *out_data, uint8_t *key, uint32_t size)
{
	uint32_t numblock = size / 16;
	uint32_t num_bytes = size;
	uint8_t rkey[176];
	uint32_t *ddata;
	uint32_t *drkey;

	expand_key(key, rkey);
	hipMalloc(&ddata, sizeof(uint8_t) * num_bytes);
	hipMalloc(&drkey, sizeof(uint8_t) * 176);
	hipMemcpy(ddata, (uint32_t *)data, sizeof(uint8_t) * num_bytes, hipMemcpyHostToDevice);
	hipMemcpy(drkey, (uint32_t *)rkey, sizeof(uint8_t) * 176, hipMemcpyHostToDevice);


	encrypt_one_block<<<(numblock + 31)/32, 32>>>((uint8_t *)ddata, (uint8_t *)drkey, numblock);
	hipDeviceSynchronize();

	hipMemcpy(out_data, ddata, sizeof(uint8_t) * num_bytes, hipMemcpyDeviceToHost);
	//check for errors
	hipError_t errCode = hipPeekAtLastError();
	if(errCode != hipSuccess){
	fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n", errCode, hipGetErrorString(errCode));
	}
	hipFree(ddata);
	hipFree(drkey);
}

void decrypt_cuda(uint8_t *data, uint8_t *out_data, uint8_t *key, uint32_t size)
{
	uint32_t numblock = size / 16;
	uint32_t num_bytes = size;
	uint8_t rkey[176];
	uint32_t *ddata;
	uint32_t *drkey;

	expand_key(key, rkey);
	hipMalloc(&ddata, sizeof(uint8_t) * num_bytes);
	hipMalloc(&drkey, sizeof(uint8_t) * 176);
	hipMemcpy(ddata, (uint32_t *)data, sizeof(uint8_t) * num_bytes, hipMemcpyHostToDevice);
	hipMemcpy(drkey, (uint32_t *)rkey, sizeof(uint8_t) * 176, hipMemcpyHostToDevice);


	decrypt_one_block<<<(numblock + 31)/32, 32>>>((uint8_t *)ddata, (uint8_t *)drkey, numblock);
	hipDeviceSynchronize();

	hipMemcpy(out_data, ddata, sizeof(uint8_t) * num_bytes, hipMemcpyDeviceToHost);
	//check for errors
	hipError_t errCode = hipPeekAtLastError();
	if(errCode != hipSuccess){
	fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n", errCode, hipGetErrorString(errCode));
	}
	hipFree(ddata);
	hipFree(drkey);
}

void print_block_hex(const uint8_t *text)
{
	for(int i = 0; i < 16; ++i){
		printf("0x%x ", text[i]);
	}
	printf("\n");
}

void write_file(char *file)
{
	uint8_t plaintext[] = { 0x00, 0x11, 0x22, 0x33, 0x44, 0x55, 0x66, 0x77, 0x88, 0x99, 0xaa, 0xbb, 0xcc, 0xdd, 0xee ,0xff
		, 0x00, 0x11, 0x22, 0x33, 0x44, 0x55, 0x66, 0x77, 0x88, 0x99, 0xaa, 0xbb, 0xcc, 0xdd, 0xee ,0xff};
	FILE *fp;
	fp = fopen(file, "wb+");
	if(!fp) {
		fprintf(stderr, "FAILED: to open plaintext!\n");
		exit(-1);
	}
	fwrite(plaintext, 1, sizeof(plaintext), fp);
	fclose(fp);
}

//read data from file
uint8_t *file_buf(const char *file, long int *size){
	int fd = open(file, O_RDONLY);
	struct stat stats;
	if(fd < 0) {
		fprintf(stderr, "Error opening file\n");
		exit(1);
	}
  	if(fstat(fd, &stats) < 0) {
  		fprintf(stderr, "Error opening file\n");
  		exit(1);
  	}
  	uint8_t *mem = (uint8_t *)mmap(NULL, stats.st_size, PROT_READ, MAP_PRIVATE, fd, 0); 
  	if(mem == MAP_FAILED) {
  		fprintf(stderr, "mmap failed\n");
  		exit(1);
  	}
  	*size = stats.st_size;
  	return mem;
}


/*
Plaintext:  00112233445566778899aabbccddeeff
Cipher key: 000102030405060708090a0b0c0d0e0f
Ciphertext: 69c4e0d86a7b0430d8cdb78070b4c55a
*/
void aes_128_testcase_single()
{
	uint8_t plaintext[16] = { 0x00, 0x11, 0x22, 0x33, 0x44, 0x55, 0x66, 0x77, 0x88, 0x99, 0xaa, 0xbb, 0xcc, 0xdd, 0xee ,0xff};
	uint8_t ciphertext[16] = {0x69, 0xc4, 0xe0, 0xd8, 0x6a, 0x7b, 0x04, 0x30, 0xd8, 0xcd, 0xb7, 0x80, 0x70, 0xb4, 0xc5, 0x5a};
	uint8_t key[16] = {0x00, 0x01, 0x02, 0x03, 0x04, 0x05, 0x06, 0x07, 0x08, 0x09, 0x0a, 0x0b, 0x0c, 0x0d, 0x0e, 0x0f};
	long int size = sizeof(plaintext);
	uint8_t *output_gpu = (uint8_t *) malloc(sizeof(uint8_t) * size);

	printf("Plaintext: \n");
	for(int i=0; i<size/16; i++) {
		print_block_hex(plaintext+16*i);
	}
	
	encrypt_cuda(plaintext, output_gpu, key, size);
	printf("Output: \n");
	for(int i=0; i<size/16; i++) {
		print_block_hex(output_gpu+16*i);
	}

	decrypt_cuda(ciphertext, output_gpu, key, size);
	printf("Output: \n");
	for(int i=0; i<size/16; i++) {
		print_block_hex(output_gpu+16*i);
	}
}

void aes_128_testcase_file()
{
	long int size;
	uint8_t key[16] = {0x00, 0x01, 0x02, 0x03, 0x04, 0x05, 0x06, 0x07, 0x08, 0x09, 0x0a, 0x0b, 0x0c, 0x0d, 0x0e, 0x0f};
	uint8_t *plaintext = file_buf(file_path, &size);
	uint8_t *output_gpu = (uint8_t *) malloc(sizeof(uint8_t) * size);
	
	printf("Plaintext: \n");
	for(int i=0; i<size/16; i++) {
		print_block_hex(plaintext+16*i);
	}
	
	encrypt_cuda(plaintext, output_gpu, key, size);
	printf("Output: \n");
	for(int i=0; i<size/16; i++) {
		print_block_hex(output_gpu+16*i);
	}

	decrypt_cuda(output_gpu, output_gpu, key, size);
	printf("Output: \n");
	for(int i=0; i<size/16; i++) {
		print_block_hex(output_gpu+16*i);
	}
}

void aes_128_testcase_1M()
{
	long int size;
	char *file = "myfile1M";
	uint8_t key[16] = {0x00, 0x01, 0x02, 0x03, 0x04, 0x05, 0x06, 0x07, 0x08, 0x09, 0x0a, 0x0b, 0x0c, 0x0d, 0x0e, 0x0f};
	uint8_t *plaintext = file_buf(file, &size);
	uint8_t *output_gpu = (uint8_t *) malloc(sizeof(uint8_t) * size);
	
	printf("Plaintext first 16 bytes: \n");
	print_block_hex(plaintext);
	
	encrypt_cuda(plaintext, output_gpu, key, size);

	decrypt_cuda(output_gpu, output_gpu, key, size);
	printf("Output first 16 bytes: \n");
	print_block_hex(output_gpu);
}

void aes_128_testcase_128M()
{
	long int size;
	char *file = "myfile128M";
	uint8_t key[16] = {0x00, 0x01, 0x02, 0x03, 0x04, 0x05, 0x06, 0x07, 0x08, 0x09, 0x0a, 0x0b, 0x0c, 0x0d, 0x0e, 0x0f};
	uint8_t *plaintext = file_buf(file, &size);
	uint8_t *output_gpu = (uint8_t *) malloc(sizeof(uint8_t) * size);
	
	printf("Plaintext first 16 bytes: \n");
	print_block_hex(plaintext);
	
	encrypt_cuda(plaintext, output_gpu, key, size);

	decrypt_cuda(output_gpu, output_gpu, key, size);
	printf("Output first 16 bytes: \n");
	print_block_hex(output_gpu);
}


int main()
{
	// write_file(file_path);
	// return 0;
	/*
	uint8_t key[16] = {0x7E, 0x24, 0x06, 0x78, 0x17, 0xFA, 0xE0, 0xD7, 0x43, 0xD6, 0xCE, 0x1F, 0x32, 0x53, 0x91, 0x63};
	uint8_t rseed[16] = {0x00, 0x6C, 0xB6, 0xDB, 0xC0, 0x54, 0x3B, 0x59, 0xDA, 0x48, 0xD9, 0x0B, 0, 0, 0, 0};
	uint8_t plaintext[33] = "\x00\x01\x02\x03\x04\x05\x06\x07\x08\x09\x0A\x0B\x0C\x0D\x0E\x0F\x10\x11\x12\x13\x14\x15\x16\x17\x18\x19\x1A\x1B\x1C\x1D\x1E\x1F";
	long int size = 32;
	*/

	// aes_128_testcase_single();
	aes_128_testcase_128M();


	/*//Test correctness
	int sum = 0;
	for(uint32_t i = 0; i < size; ++i){
		sum += abs(ciphertext[i] - output_gpu[i]);
	}
	printf("Sum = %d  (should be zero if correct)\n", sum);*/
	return 0;
}